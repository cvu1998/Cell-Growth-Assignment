#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <Elysium.h>

__global__ void calculatePosition(Elysium::Vector2* positions, Elysium::Vector2 offset, float size,
    unsigned int numberOfCellsX, unsigned int numberOfCells)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < numberOfCells)
    {
        positions[i].x = ((float)(i % numberOfCellsX) - offset.x) * size;
        positions[i].y = ((float)(i / numberOfCellsX) - offset.y) * size;
    }
}